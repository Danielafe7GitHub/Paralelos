#include "hip/hip_runtime.h"
#include <stdio.h>
#include <>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <Windows.h>
#include <iostream>
#define TILE_WIDTH 16
using namespace std;
__global__
void vecMultKernel(float* A, float* B, float* C, int n){
	int i = threadIdx.x + blockDim.x * blockIdx.x;
	if (i<n){
		C[i] = 0;
		for (int j = 0; j<n; j++)
			C[i] += A[j*n + i] * B[j];
	}
}
__global__ void MatrixMultKernel(float* d_M, float* d_N, float* d_P, int width){
	__shared__ float Mds[TILE_WIDTH][TILE_WIDTH];
	__shared__ float Nds[TILE_WIDTH][TILE_WIDTH];
	int bx = blockIdx.x; int by = blockIdx.y;
	int tx = threadIdx.x; int ty = threadIdx.y;
	// Identify the row and column of the d_P element to work on
	int Row = by * TILE_WIDTH + ty;
	int Col = bx * TILE_WIDTH + tx;
	float Pvalue = 0;
	// Loop over the d_M and d_N tiles required to compute d_P element
	for (int m = 0; m < Width/TILE_WIDTH; ++m) {
	// Coolaborative loading of d_M and d_N tiles into shared memory
		Mds[ty][tx] = d_M[Row*Width + m*TILE_WIDTH + tx];
		Nds[ty][tx] = d_N[(m*TILE_WIDTH + ty)*Width + Col];
		__syncthreads();
		for (int k = 0; k < TILE_WIDTH; ++k) {
			Pvalue += Mds[ty][k] * Nds[k][tx];
		}
		__syncthreads();
	}
	d_P[Row*Width + Col] = Pvalue; 
}
void matrizXvector(float* A, float* B, float* C, int n) {
	int size = n * n * sizeof(float);
	int sizevect = n * sizeof(float);
	float *d_A, *d_B, *d_C;
	///Redimensionar y copiar de Host a Device
	hipMalloc((void **)&d_A, size);
	hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);
	hipMalloc((void **)&d_B, sizevect);
	hipMemcpy(d_B, B, sizevect, hipMemcpyHostToDevice);
	hipMalloc((void **)&d_C, sizevect);

	//Llamada Kernel
	vecMultKernel <<< ceil((n*n) / 256.0), 256 >>> (d_A, d_B, d_C, n);
	//ceil se asegura de que tener suficientes hilos para cubrir los elementos

	//copiar de Device a Host
	hipMemcpy(C, d_C, sizevect, hipMemcpyDeviceToHost);

	//liberar memoria
	hipFree(d_A); hipFree(d_B); hipFree(d_C);
}
void matrizXmatriz(float* A, float* B, float* C, int n) {
	int size = n * n * sizeof(float);
	int sizevect = n * n * sizeof(float);
	float *d_A, *d_B, *d_C;
	///Redimensionar y copiar de Host a Device
	hipMalloc((void **)&d_A, size);
	hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);
	hipMalloc((void **)&d_B, size);
	hipMemcpy(d_B, B, sizevect, hipMemcpyHostToDevice);
	hipMalloc((void **)&d_C, size);

	//Llamada Kernel
	MatrixMultKernel <<< ceil((n*n) / 256.0), 256 >>> (d_A, d_B, d_C, n);
	//ceil se asegura de que tener suficientes hilos para cubrir los elementos

	//copiar de Device a Host
	hipMemcpy(C, d_C, size, hipMemcpyDeviceToHost);

	//liberar memoria
	hipFree(d_A); hipFree(d_B); hipFree(d_C);
}
void mostrarM(float* matriz, float fila, float columna)
{
	for (int x = 0; x < fila; x++)
	{
		for (int y = 0; y < columna; y++)
		{
			int puesto = x*columna + y;
			printf("%3.0f ", matriz[puesto]);
		}
		printf("\n");
	}
}
int main() {
	int fila , columna ;
	cout<<"ingrese dimensiones"<< endl;
	cin>>fila;
	columna=fila;
	float* A = (float*)malloc(fila*columna*sizeof(float));
	float* B = (float*)malloc(fila*columna*sizeof(float));
	float* C = (float*)malloc(fila*columna*sizeof(float));
	for (int i = 0; i < fila*columna; i++)
	{
		A[i] = i;
	}
	for (int i = 0; i < fila*columna; i++)
	{
		B[i] = i;
	}

	cout<<" matriz B "<<endl;
	mostrarM(B, fila, columna);
	cout<<" * matriz A "<<endl;
	mostrarM(A, fila, columna);
	
	cout <<"Resultado"<<endl;
	//matrizXvector(A, B, C, fila);
	matrizXmatriz(A, B, C, fila);
	mostrarM(C, fila , columna);

	system("PAUSE");
	exit(0);
}
