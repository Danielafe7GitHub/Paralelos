#include "hip/hip_runtime.h"
#include <stdio.h>
#include <>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <Windows.h>
#include <iostream>
using namespace std;
#define BLOCK 8
__global__
void matrizXescalarK(float* A, float B, float* C, int f,int c,int e){

	int fila = blockIdx.y*blockDim.y + threadIdx.y;
	int columna = blockIdx.x*blockDim.x + threadIdx.x;
	int profundidad = blockIdx.z*blockDim.z + threadIdx.z;

	if ((fila < f) && (columna < c) && (profundidad < e)) {
		C[fila*c + columna + profundidad*c*f] = B * A[fila*c + columna+profundidad*c*f];
	}
}
void matrizXescalar(float* A, float B, float* C, int f,int c,int e) {
	int size = f * c * e *  sizeof(float);
	float *d_A, *d_C;
	hipMalloc((void **)&d_A, size);
	hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);
	hipMalloc((void **)&d_C, size);
	///dividir las filas, columnas y tercera dimension en bloques
	int NumBlocks1 = c / BLOCK;
	int NumBlocks2 = f / BLOCK;
	int NumBlocks3 = e / BLOCK;

	///cubrir todos los elementos
	if (c % BLOCK) NumBlocks1++;
	if (f % BLOCK) NumBlocks2++;
	if (e % BLOCK) NumBlocks3++;

	dim3 dime1(NumBlocks1, NumBlocks2, NumBlocks3);
	dim3 dime2(BLOCK, BLOCK, BLOCK);
	matrizXescalarK << < dime1, dime2 >> > (d_A, B, d_C, f, c,e);

	/// Copiar de Device a Host
	hipMemcpy(C, d_C, size, hipMemcpyDeviceToHost);
	/// Liberar Memoria
	hipFree(d_A); hipFree(d_C);
}
void mostrarM(float* matriz, float fila, float columna)
{
	for (int x = 0; x < fila; x++)
	{
		for (int y = 0; y < columna; y++)
		{
			int puesto = x*columna + y;
			printf("%3.0f ", matriz[puesto]);
		}
		printf("\n");
	}
}
int main() {
	int fila, columna, profundidad;
	cout<<"ingrese dimensiones"<<endl;
	cin>>fila;
	columna=profundidad=fila;
	float* A = (float*)malloc(fila*columna*profundidad*sizeof(float));
	float B;
	cout<<"ingrese escalar"<<endl;
	cin>>B;
	float* C = (float*)malloc(fila*columna*profundidad*sizeof(float));
	for (int i = 0; i < fila*columna*profundidad; i++)
	{
		A[i] = i;
	}
	cout<<"Matriz"<<endl;
	mostrarM(A, fila, columna*profundidad);
	cout<<" * "<<B<<" = "<<endl;
	matrizXescalar(A, B, C, fila,columna,profundidad);
	mostrarM(C, fila, columna*profundidad);

	system("PAUSE");
	exit(0);
}
