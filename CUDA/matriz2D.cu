#include "hip/hip_runtime.h"
#include <stdio.h>
#include <>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <Windows.h>
#include <iostream>
using namespace std;
#define BLOCK 16
__global__
void matrizXescalar1(float* A, float B, float* C, int f,int c){
	int fila = blockIdx.y*blockDim.y + threadIdx.y;
	int columna = blockIdx.x*blockDim.x + threadIdx.x;
	if ((fila < f) && (columna < c)) {
		C[fila*c + columna] = B * A[fila*c + columna];
	}
}
void matrizXescalar(float* A, float B, float* C, int f,int c) {
	int size = f * c * sizeof(float);
	float *d_A, *d_C;
	///Redimensionar
	hipMalloc((void **)&d_A, size);
	hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);
	hipMalloc((void **)&d_C, size);

	///dividir las filas y columnas en bloques
	int NumBlocks1 = c / BLOCK;
	int NumBlocks2 = f / BLOCK;
	
	///cubrir todos los elementos
	if (c % BLOCK) NumBlocks1++;
	if (f%BLOCK) NumBlocks2++;


	dim3 dime1(NumBlocks1, NumBlocks2);///bloques
	dim3 dime2(BLOCK, BLOCK);		   /// hilos
	matrizXescalar1 << < dime1, dime2 >> > (d_A, B, d_C, f, c);

	/// Copiar de Device a Host
	hipMemcpy(C, d_C, size, hipMemcpyDeviceToHost);
	/// Liberar Memoria
	hipFree(d_A); hipFree(d_C);
}
void mostrarM(float* matriz, float fila, float columna)
{
	for (int x = 0; x < fila; x++)
	{
		for (int y = 0; y < columna; y++)
		{
			int puesto = x*columna + y;
			printf("%3.0f ", matriz[puesto]);
		}
		printf("\n");
	}
}
int main() {
	cout<<"ingrese dimensiones "<<endl;
	int fila,columna;
	cout<<"filas:"<<endl;
	cin>>fila;
	cout<<"columnas:"<<endl;
	cin>>columna;

	float* A = (float*)malloc(fila*columna*sizeof(float));
	float B;
	cout<<"ingrese escalar"<<endl;
	cin>>B;
	float* C = (float*)malloc(fila*columna*sizeof(float));
	for (int i = 0; i < fila*columna; i++)
	{
		A[i] = i;
	}
	cout<<"Matriz"<<endl;
	mostrarM(A, fila, columna);
	cout<<" * "<<B<<" = "<<endl;
	matrizXescalar(A, B, C, fila,columna);
	mostrarM(C, fila, columna);
	
	
	system("PAUSE");
	exit(0);
}
